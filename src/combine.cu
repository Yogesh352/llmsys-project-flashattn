#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <sstream>
#include <fstream>

#define BLOCK_DIM 1024
#define MAX_DIMS 10
#define TILE 32
#define BASE_THREAD_NUM 32

#define ADD_FUNC       1
#define MUL_FUNC       2
#define ID_FUNC        3
#define NEG_FUNC       4
#define LT_FUNC        5
#define EQ_FUNC        6
#define SIGMOID_FUNC   7
#define RELU_FUNC      8
#define RELU_BACK_FUNC 9
#define LOG_FUNC       10
#define LOG_BACK_FUNC  11
#define EXP_FUNC       12
#define INV_FUNC       13
#define INV_BACK_FUNC  14
#define IS_CLOSE_FUNC  15
#define MAX_FUNC       16
#define POW            17
#define TANH           18

__device__ float fn(int fn_id, float x, float y=0) {
    switch(fn_id) {
      case ADD_FUNC: {
        return x + y;
      }
      case MUL_FUNC: {
        return x * y;
      }
      case ID_FUNC: {
      	return x;
      }
      case NEG_FUNC: {
        return -x;
      }
      case LT_FUNC: {
        if (x < y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case EQ_FUNC: {
        if (x == y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case SIGMOID_FUNC: {
        if (x >= 0) {
          return 1.0 / (1.0 + exp(-x));
        }
        else {
          return exp(x) / (1.0 + exp(x));
        }
      }
      case RELU_FUNC: {
        return max(x, 0.0);
      }
      case RELU_BACK_FUNC: {
        if (x > 0) {
          return y;
        }
        else {
          return 0.0;
        }
      }
      case LOG_FUNC: {
        return log(x + 1e-6);
      }
      case LOG_BACK_FUNC: {
        return y / (x + 1e-6);
      }
      case EXP_FUNC: {
        return exp(x);
      }
      case INV_FUNC: {
        return float(1.0 / x);
      }
      case INV_BACK_FUNC: {
        return -(1.0 / (x * x)) * y;
      }
      case IS_CLOSE_FUNC: {
        return (x - y < 1e-2) && (y - x < 1e-2);
      }
      case MAX_FUNC: {
        if (x > y) {
          return x;
        }
        else {
          return y;
        }
      }
      case POW: {
        return pow(x, y);
      }
      case TANH: {
        return tanh(x);
      }
      default: {
        return x + y;
      }
    }
    
}


__device__ int index_to_position(const int* index, const int* strides, int num_dims) {
    int position = 0;
    for (int i = 0; i < num_dims; ++i) {
        position += index[i] * strides[i];
    }
    return position;
}

__device__ void to_index(int ordinal, const int* shape, int* out_index, int num_dims) {
    int cur_ord = ordinal;
    for (int i = num_dims - 1; i >= 0; --i) {
        int sh = shape[i];
        out_index[i] = cur_ord % sh;
        cur_ord /= sh;
    }
}

__device__ void broadcast_index(const int* big_index, const int* big_shape, const int* shape, int* out_index, int num_dims_big, int num_dims) {
    for (int i = 0; i < num_dims; ++i) {
        if (shape[i] > 1) {
            out_index[i] = big_index[i + (num_dims_big - num_dims)];
        } else {
            out_index[i] = 0;
        }
    }
}


__global__ void MatrixMultiplyKernel(
    float* out,
    const int* out_shape,
    const int* out_strides,
    float* a_storage,
    const int* a_shape,
    const int* a_strides,
    float* b_storage,
    const int* b_shape,
    const int* b_strides
) {

     __shared__ float a_shared[TILE][TILE];
    __shared__ float b_shared[TILE][TILE];

    
    int batch = blockIdx.z;
    int a_batch_stride = a_shape[0] > 1 ? a_strides[0] : 0;
    int b_batch_stride = b_shape[0] > 1 ? b_strides[0] : 0;

        
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = bx * blockDim.x + tx;
    int col = by * blockDim.y + ty;

    int height = out_shape[1];
    int width = out_shape[2];

    float PValue = 0;
    int p = 0;

    for (int ph = 0; ph < a_shape[2]; ph += TILE) {
        p = ty + ph;
        if (row < a_shape[1] && p < a_shape[2]){
          a_shared[tx][ty] = a_storage[a_batch_stride * batch +a_strides[1] * row + a_strides[2] * p];
        } else {
          a_shared[tx][ty] = 0.0f; 
        }
        p = tx + ph;

        if (col < b_shape[2] && p < b_shape[1]){
          b_shared[tx][ty] = b_storage[b_batch_stride * batch + b_strides[1] * p + b_strides[2] * col];
        } else{
          b_shared[tx][ty] = 0.0f; 
        }

        __syncthreads();

        for (int k = 0; k < TILE; ++k){
            PValue +=  a_shared[tx][k] * b_shared[k][ty];   
        }
        __syncthreads();

    }

    if (row < height && col < width) {
        out[out_strides[0] * batch + out_strides[1] * row + out_strides[2] * col] = PValue;
    }
}


__global__ void mapKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int shape_size,
    int fn_id
) {
    int out_index[MAX_DIMS];
    int in_index[MAX_DIMS];
    
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < out_size){
      to_index(idx, out_shape, out_index, shape_size);
      broadcast_index(out_index, out_shape, in_shape, in_index, shape_size, shape_size);

      int in_position = index_to_position(in_index, in_strides, shape_size);
      int out_position = index_to_position(out_index, out_strides, shape_size);

      out[out_position] = fn(fn_id, in_storage[in_position]);
    }
}


__global__ void reduceKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim,
    float reduce_value,
    int shape_size,
    int fn_id
) {
    __shared__ double cache[BLOCK_DIM];
    int out_index[MAX_DIMS];
    int in_index[MAX_DIMS];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    to_index(idx, out_shape, out_index, shape_size);
    broadcast_index(out_index, out_shape, a_shape, in_index, shape_size, shape_size);
    int out_position = index_to_position(out_index, out_strides, shape_size);

    int tid = threadIdx.x;
    int in_pos = index_to_position(in_index, a_strides, shape_size);
    
    cache[tid] = fn(fn_id, reduce_value, a_storage[in_pos]);
    for (long s = 1; s < a_shape[reduce_dim]; s +=1){
      in_index[reduce_dim] += 1;
      in_pos = index_to_position(in_index, a_strides, shape_size);
      cache[tid] = fn(fn_id, cache[tid], a_storage[in_pos]);
    }

    out[out_position] = cache[tid];
}

__global__ void zipKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_shape_size,
    int fn_id
) {
    int out_index[MAX_DIMS];
    int a_index[MAX_DIMS];
    int b_index[MAX_DIMS];

   
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < out_size){
      to_index(idx, out_shape, out_index, out_shape_size);
      int out_position = index_to_position(out_index, out_strides, out_shape_size);

      broadcast_index(out_index, out_shape, a_shape, a_index, out_shape_size, a_shape_size);
      int a_pos = index_to_position(a_index, a_strides, a_shape_size);
      
      broadcast_index(out_index, out_shape, b_shape, b_index, out_shape_size, b_shape_size);
      int b_pos = index_to_position(b_index, b_strides, b_shape_size);
      out[out_position] = fn(fn_id, a_storage[a_pos], b_storage[b_pos]);

    }
}


extern "C" {

void MatrixMultiply(
    float* out,
    int* out_shape,
    int* out_strides,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int batch, int m, int p
) {
    int n = a_shape[2];

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, batch * m * n * sizeof(float));
    hipMalloc(&d_b, batch * n * p * sizeof(float));
    hipMalloc(&d_out, batch * m * p * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, 3 * sizeof(int));
    hipMalloc(&d_out_strides, 3 * sizeof(int));
    hipMalloc(&d_a_shape, 3 * sizeof(int));
    hipMalloc(&d_a_strides, 3 * sizeof(int));
    hipMalloc(&d_b_shape, 3 * sizeof(int));
    hipMalloc(&d_b_strides, 3 * sizeof(int));


    // Copy data to the device
    hipMemcpy(d_a, a_storage, batch * m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, batch * n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, 3 * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = BASE_THREAD_NUM;
    dim3 blockDims(threadsPerBlock, threadsPerBlock, 1); // Adjust these values based on your specific requirements
    dim3 gridDims((m + threadsPerBlock - 1) / threadsPerBlock, (p + threadsPerBlock - 1) / threadsPerBlock, batch);
    MatrixMultiplyKernel<<<gridDims, blockDims>>>(
        d_out, d_out_shape, d_out_strides, d_a, d_a_shape, d_a_strides, d_b, d_b_shape, d_b_strides
    );

    // Copy back to the host
    hipMemcpy(out, d_out, batch * m * p * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Matmul Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}

void tensorMap(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int in_size,
    int shape_size,
    int fn_id
) {

    float *d_out, *d_in;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_in, in_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_in_shape, *d_in_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_in_shape, shape_size * sizeof(int));
    hipMalloc(&d_in_strides, shape_size * sizeof(int));

    hipMemcpy(d_in, in_storage, in_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_shape, in_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_strides, in_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    mapKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, 
      d_in, d_in_shape, d_in_strides, 
      shape_size, fn_id);
    
    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Map Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_in_shape);
    hipFree(d_in_strides);
}


void tensorZip(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_size,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_size,
    int b_shape_size,
    int fn_id
) {

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc((void **)&d_a, a_size * sizeof(float));
    hipMalloc(&d_b, b_size * sizeof(float));
    hipMalloc(&d_out, out_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, out_shape_size * sizeof(int));
    hipMalloc(&d_out_strides, out_shape_size * sizeof(int));
    hipMalloc(&d_a_shape, a_shape_size * sizeof(int));
    hipMalloc(&d_a_strides, a_shape_size * sizeof(int));
    hipMalloc(&d_b_shape, b_shape_size * sizeof(int));
    hipMalloc(&d_b_strides, b_shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, b_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, b_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, b_shape_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    zipKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, out_shape_size,
      d_a, d_a_shape, d_a_strides, a_shape_size,
      d_b, d_b_shape, d_b_strides, b_shape_size,
      fn_id);

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();


    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Zip Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}



void tensorReduce(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim, 
    float reduce_value,
    int shape_size,
    int fn_id
) {
    int a_size = out_size * a_shape[reduce_dim];
    float *d_out, *d_a;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_a, a_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_a_shape, shape_size * sizeof(int));
    hipMalloc(&d_a_strides, shape_size * sizeof(int));

    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    reduceKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_out, d_out_shape, d_out_strides, out_size, 
        d_a, d_a_shape, d_a_strides, 
        reduce_dim, reduce_value, shape_size, fn_id
    );

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Reduce Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
}

}