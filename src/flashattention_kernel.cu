#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstddef>
#include "includes/kernels.h"
#include "includes/cuda_util.h"

#include <hip/hip_cooperative_groups.h>

__global__ void forward_kernel(const float *Q, const float *K, const float *V, const int N, const int d,
                               const int num_tiles_K, const int num_tiles_Q, const int block_size_K, const int block_size_Q,
                               const float softmax_scale, float *l, float *m, float *O)
{
    int thread_idx = threadIdx.x;
    int batch_idx = blockIdx.x;
    int head_idx = blockIdx.y;

    int qkv_offset = (batch_idx * gridDim.y * N * d) + (head_idx * N * d);
    // int qkv_offset_2 = (batch_idx * gridDim.y * N * d) + (head_idx / gridDim.y * (block_size_K)*d) + (head_idx * d);
    // int qkv_offset_2 = (batch_idx * gridDim.y * N * d) + (block_size_K * thread_idx * gridDim.y * d) + (head_idx * d);

    // Offset Calculation
    // Flattened 1D Data Shape: (B, N, nh, d)
    // gridDim shape = (x, y) = (B, nh)
    // 1) Batch -> (batch_idx * gridDim.y * N * d)
    // 2) N -> (block_size_K * thread_idx * gridDim.y * d)
    // 3) nh -> head_idx * d

    int lm_offset = (batch_idx * gridDim.y * N) + (head_idx * N);

    extern __shared__ float shared_memory[];
    int tile_size = block_size_K * d;
    float *Qi = shared_memory;
    float *Kj = &shared_memory[tile_size];
    float *Vj = &shared_memory[tile_size * 2];
    float *S = &shared_memory[tile_size * 3];

    for (int tile_idx_K = 0; tile_idx_K < num_tiles_K; tile_idx_K++)
    {
        for (int x = 0; x < d; x++)
        {

            Kj[(thread_idx * d) + x] = K[qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x];
            Vj[(thread_idx * d) + x] = V[qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x];
            // printf("Head %d: Key[%d] = %f, Value[%d] = %f, qkv_offset = %d\n", head_idx, qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x, Kj[(thread_idx * d) + x],
            // qkv_offset + (tile_size * tile_idx_K) + (thread_idx * gridDim.y * d) + x, Vj[(thread_idx * d) + x], qkv_offset);
            // printf("ADDITIONAL ON TOP OF OFFSET = %d\n",   (thread_idx * gridDim.y * d));

            // printf("K and V access: %d\n", qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x);
        }
        __syncthreads();

        for (int tile_idx_Q = 0; tile_idx_Q < num_tiles_Q; tile_idx_Q++)
        {

            for (int x = 0; x < d; x++)
            {
                Qi[(thread_idx * d) + x] = Q[qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x];
                // printf("Head %d: Query[%d] = %f\n", head_idx, qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x,  Qi[(thread_idx * d) + x]);

                // printf("Q access: %d\n", qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x);
            }

            float prev_m = m[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx];
            float prev_l = l[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx];
            // printf("m and l access: %d\n", lm_offset + (block_size_Q * tile_idx_Q) + thread_idx);

            float row_max = -INFINITY;
            for (int y = 0; y < block_size_K; y++)
            {
                float sum = 0;
                for (int x = 0; x < d; x++)
                {
                    sum += Qi[(thread_idx * d) + x] * Kj[(y * d) + x];
                    // printf("For index: %d, Head %d: Query[%d] = %f, Kj[%d] = %f\n", (block_size_K * thread_idx) + y, head_idx, (thread_idx * d) + x,  Qi[(thread_idx * d) + x], (y * d) + x,  Kj[(y * d) + x]);
                }
                // printf("sum = %f\n", sum);
                sum *= softmax_scale;
                // printf("sum after scaling = %f\n", sum);
                S[(block_size_K * thread_idx) + y] = sum;
                // printf("Head %d: S[%d] = %f\n",  head_idx, (block_size_K * thread_idx) + y, S[(block_size_K * thread_idx) + y] );

                if (sum > row_max)
                    row_max = sum;
            }

            float row_sum = 0;
            for (int y = 0; y < block_size_K; y++)
            {
                S[(block_size_K * thread_idx) + y] = __expf(S[(block_size_K * thread_idx) + y] - row_max);
                row_sum += S[(block_size_K * thread_idx) + y];
            }

            float new_m = fmax(prev_m, row_max);
            float new_l = (__expf(prev_m - new_m) * prev_l) + (__expf(row_max - new_m) * row_sum);

            for (int x = 0; x < d; x++)
            {
                float weighted_sum = 0;
                for (int y = 0; y < block_size_K; y++)
                {
                    weighted_sum += S[(block_size_K * thread_idx) + y] * Vj[(y * d) + x];
                }
                __syncthreads();
                int index = qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x;
                O[index] = (1 / new_l) *
                           ((prev_l * __expf(prev_m - new_m) * O[index]) +
                            (__expf(row_max - new_m) * weighted_sum));
                // printf("O access after update: %d\n", index);
            }

            m[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx] = new_m;
            l[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx] = new_l;
        }
        __syncthreads();
    }
}

extern "C"
{
    void launch_flashattention_forward(float *Q, float *K, float *V, float *O, float *l, float *m, int batch_size, int num_heads, int N, int d)
    {

        int block_size_K, block_size_Q;
        // printf("N: %d\n", N);
        block_size_K = min(N, 2048/d); block_size_Q = min(N, 2048/d);

        // if (d <= 64) {
        //     block_size_K = min(N, 32); block_size_Q = min(N, 32);
        // } else if (d <= 128) {
        //     block_size_K = min(N, 16); block_size_Q = min(N, 16);
        // } else if (d <= 256) {
        //     block_size_K = min(N, 8); block_size_Q = min(N, 8);
        // } else if (d <= 512) {
        //     block_size_K = min(N, 4); block_size_Q = min(N, 4);
        // }

        int max_sram_size;
        hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

        // int sram_size = max_sram_size;

        // Compute the largest power of 2 <= num
        // sram_size |= (sram_size >> 1);
        // sram_size |= (sram_size >> 2);
        // sram_size |= (sram_size >> 4);
        // sram_size |= (sram_size >> 8);
        // sram_size |= (sram_size >> 16);
        // sram_size = (sram_size + 1) >> 1;

        // sram_size /= sizeof(float);

        // Accounting for overhead
        // sram_size *= 0.7;

        // M = SRAM size
        // Block Size (Q) = M / 4d
        // Block Size (K,V) = min(M / 4d, d)

        // Set according to your GPU
        int sram_size = 8192/4;

        // block_size_K = min(sram_size / (4 * d), N);
        // block_size_Q = min(sram_size / (4 * d), d);

        const int num_tiles_K = ceil((float)N / block_size_K);
        const int num_tiles_Q = ceil((float)N / block_size_Q);

        // printf("Q[0]: %f\n", Q[0]);
        // printf("Q[1]: %f\n", Q[1]);
        // printf("Q[2]: %f\n", Q[2]);
        // printf("Q[3]: %f\n", Q[3]);
        // printf("Q[4]: %f\n", Q[4]);
        // printf("Q[5]: %f\n", Q[5]);
        // printf("Q[6]: %f\n", Q[6]);
        // printf("Q[7]: %f\n", Q[7]);

        const float softmax_scale = 1.0 / sqrt(d);

        float *d_Q, *d_K, *d_V, *d_O, *d_l, *d_m;
        const int Q_size = batch_size * num_heads * N * d;
        const int l_size = batch_size * num_heads * N;

        hipMalloc((void **)&d_Q, Q_size * sizeof(float));
        hipMalloc((void **)&d_K, Q_size * sizeof(float));
        hipMalloc((void **)&d_V, Q_size * sizeof(float));
        hipMalloc((void **)&d_O, Q_size * sizeof(float));
        hipMalloc((void **)&d_l, l_size * sizeof(float));
        hipMalloc((void **)&d_m, l_size * sizeof(float));

        hipMemcpy(d_Q, Q, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_K, K, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_V, V, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_O, O, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_l, l, l_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_m, m, l_size * sizeof(float), hipMemcpyHostToDevice);

        const int shared_mem_size = (3 * block_size_K * d * sizeof(float)) + (block_size_K * block_size_Q * sizeof(float));

        printf("d: %d | num_head: %d | N: %d | Block Size Q: %d | Block Size K: %d | Max shared memory: %d, requested shared memory: %d \n", d, num_heads, N, block_size_Q, block_size_K, max_sram_size, shared_mem_size);

        // if (shared_mem_size >= max_sram_size) {
        //     fprintf(stderr, "Too much SRAM requested: %f\n", shared_mem_size);
        //     exit(EXIT_FAILURE);
        // }

        dim3 grid_dim(batch_size, num_heads);
        dim3 block_dim(block_size_K);

        // Launch the kernel
        forward_kernel<<<grid_dim, block_dim, shared_mem_size>>>(
            d_Q, d_K, d_V, N, d, num_tiles_K, num_tiles_Q, block_size_K, block_size_Q, softmax_scale, d_l, d_m, d_O);
        hipDeviceSynchronize();

        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Error in kernel launch: %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        hipMemcpy(O, d_O, Q_size * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_Q);
        hipFree(d_K);
        hipFree(d_V);
        hipFree(d_O);
        hipFree(d_l);
        hipFree(d_m);
    }
}

__global__ void forward_kernel_causal(const float *Q, const float *K, const float *V, const int N, const int d,
                                      const int num_tiles_K, const int num_tiles_Q, const int block_size_K, const int block_size_Q,
                                      const float softmax_scale, float *l, float *m, float *O)
{

    int thread_idx = threadIdx.x;
    int batch_idx = blockIdx.x;
    int head_idx = blockIdx.y;

    int qkv_offset = (batch_idx * gridDim.y * N * d) + (head_idx * N * d);
    int lm_offset = (batch_idx * gridDim.y * N) + (head_idx * N);

    extern __shared__ float shared_memory[];
    int tile_size = block_size_K * d;
    float *Qi = shared_memory;
    float *Kj = &shared_memory[tile_size];
    float *Vj = &shared_memory[tile_size * 2];
    float *S = &shared_memory[tile_size * 3];

    for (int tile_idx_K = 0; tile_idx_K < num_tiles_K; tile_idx_K++)
    {
        for (int x = 0; x < d; x++)
        {
            Kj[(thread_idx * d) + x] = K[qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x];
            Vj[(thread_idx * d) + x] = V[qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x];
        }
        __syncthreads();

        for (int tile_idx_Q = tile_idx_K; tile_idx_Q < num_tiles_Q; tile_idx_Q++)
        {
            if (tile_idx_Q * block_size_Q + thread_idx >= N)
                break;

            for (int x = 0; x < d; x++)
            {
                Qi[(thread_idx * d) + x] = Q[qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x];
            }

            float prev_m = m[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx];
            float prev_l = l[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx];

            float row_max = -INFINITY;
            for (int y = 0; y < block_size_K; y++)
            {
                if (tile_idx_K * block_size_K + y >= N)
                    break;
                float sum = 0;
                for (int x = 0; x < d; x++)
                {
                    sum += Qi[(thread_idx * d) + x] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                if (tile_idx_Q * block_size_Q + thread_idx < tile_idx_K * block_size_K + y)
                    sum = -INFINITY;
                S[(block_size_K * thread_idx) + y] = sum;

                if (sum > row_max)
                    row_max = sum;
            }

            float row_sum = 0;
            for (int y = 0; y < block_size_K; y++)
            {
                if (tile_idx_K * block_size_K + y >= N)
                    break;
                if (tile_idx_Q * block_size_Q + thread_idx < tile_idx_K * block_size_K + y)
                    S[(block_size_K * thread_idx) + y] = 0;
                else
                    S[(block_size_K * thread_idx) + y] = __expf(S[(block_size_K * thread_idx) + y] - row_max);
                row_sum += S[(block_size_K * thread_idx) + y];
            }

            float new_m = fmax(prev_m, row_max);
            float new_l = (__expf(prev_m - new_m) * prev_l) + (__expf(row_max - new_m) * row_sum);

            for (int x = 0; x < d; x++)
            {
                float weighted_sum = 0;
                for (int y = 0; y < block_size_K; y++)
                {
                    if (tile_idx_K * block_size_K + y >= N)
                        break;
                    weighted_sum += S[(block_size_K * thread_idx) + y] * Vj[(y * d) + x];
                }
                O[qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x] = (1 / new_l) *
                                                                                  ((prev_l * __expf(prev_m - new_m) * O[qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x]) +
                                                                                   (__expf(row_max - new_m) * weighted_sum));
            }
            m[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx] = new_m;
            l[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx] = new_l;
        }
        __syncthreads();
    }
}

extern "C"
{
    void launch_flashattention_forward_causal(float *Q, float *K, float *V, float *O, float *l, float *m, int batch_size, int num_heads, int N, int d)
    {

        int block_size_K, block_size_Q;

        if (d <= 64)
        {
            block_size_K = min(N, 32);
            block_size_Q = min(N, 32);
        }
        else if (d <= 128)
        {
            block_size_K = min(N, 16);
            block_size_Q = min(N, 16);
        }
        else if (d <= 256)
        {
            block_size_K = min(N, 8);
            block_size_Q = min(N, 8);
        }
        else if (d <= 512)
        {
            block_size_K = min(N, 4);
            block_size_Q = min(N, 4);
        }

        const int num_tiles_K = ceil((float)N / block_size_K);
        const int num_tiles_Q = ceil((float)N / block_size_Q);

        const float softmax_scale = 1.0 / sqrt(d);

        float *d_Q, *d_K, *d_V, *d_O, *d_l, *d_m;
        const int Q_size = batch_size * num_heads * N * d;
        const int l_size = batch_size * num_heads * N;

        hipMalloc((void **)&d_Q, Q_size * sizeof(float));
        hipMalloc((void **)&d_K, Q_size * sizeof(float));
        hipMalloc((void **)&d_V, Q_size * sizeof(float));
        hipMalloc((void **)&d_O, Q_size * sizeof(float));
        hipMalloc((void **)&d_l, l_size * sizeof(float));
        hipMalloc((void **)&d_m, l_size * sizeof(float));

        // hipMemcpy(d_Q, Q, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_K, K, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_V, V, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_l, l, l_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_m, m, l_size * sizeof(float), hipMemcpyHostToDevice);

        const int shared_mem_size = (3 * block_size_K * d * sizeof(float)) + (block_size_K * block_size_Q * sizeof(float));

        dim3 grid_dim(batch_size, num_heads);
        dim3 block_dim(block_size_K);

        // Launch the kernel
        forward_kernel<<<grid_dim, block_dim, shared_mem_size>>>(
            d_Q, d_K, d_V, N, d, num_tiles_K, num_tiles_Q, block_size_K, block_size_Q, softmax_scale, d_l, d_m, d_O);
        hipDeviceSynchronize();

        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Error in kernel launch: %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        hipMemcpy(O, d_O, Q_size * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_Q);
        hipFree(d_K);
        hipFree(d_V);
        hipFree(d_O);
        hipFree(d_l);
        hipFree(d_m);
    }
}