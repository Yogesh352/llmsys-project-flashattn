#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstddef>
#include "includes/kernels.h"
#include "includes/cuda_util.h"

#include <hip/hip_cooperative_groups.h>

__global__ void forward_kernel(const float *Q, const float *K, const float *V, const int N, const int d,
                               const int num_tiles_K, const int num_tiles_Q, const int block_size_K, const int block_size_Q,
                               const float softmax_scale, float *l, float *m, float *O)
{
    int thread_idx = threadIdx.x;
    int batch_idx = blockIdx.x;
    int head_idx = blockIdx.y;

    int qkv_offset = (batch_idx * gridDim.y * N * d) + (head_idx * N * d);
    // int qkv_offset_2 = (batch_idx * gridDim.y * N * d) + (head_idx / gridDim.y * (block_size_K)*d) + (head_idx * d);
    // int qkv_offset_2 = (batch_idx * gridDim.y * N * d) + (block_size_K * thread_idx * gridDim.y * d) + (head_idx * d);

    // Offset Calculation
    // Flattened 1D Data Shape: (B, N, nh, d)
    // gridDim shape = (x, y) = (B, nh)
    // 1) Batch -> (batch_idx * gridDim.y * N * d)
    // 2) N -> (block_size_K * thread_idx * gridDim.y * d)
    // 3) nh -> head_idx * d

    int lm_offset = (batch_idx * gridDim.y * N) + (head_idx * N);

    extern __shared__ float shared_memory[];
    int tile_size = block_size_K * d;
    float *Qi = shared_memory;
    float *Kj = &shared_memory[tile_size];
    float *Vj = &shared_memory[tile_size * 2];
    // float *S = &shared_memory[tile_size * 2 + block_size_K * block_size_Q];
    float *S = &shared_memory[tile_size * 3];


    for (int tile_idx_K = 0; tile_idx_K < num_tiles_K; tile_idx_K++)
    {
        for (int x = 0; x < d; x++)
        {

            Kj[(thread_idx * d) + x] = K[qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x];
            Vj[(thread_idx * d) + x] = V[qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x];
            // printf("Head %d: Key[%d] = %f, Value[%d] = %f, qkv_offset = %d\n", head_idx, qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x, Kj[(thread_idx * d) + x],
            // qkv_offset + (tile_size * tile_idx_K) + (thread_idx * gridDim.y * d) + x, Vj[(thread_idx * d) + x], qkv_offset);
            // printf("ADDITIONAL ON TOP OF OFFSET = %d\n",   (thread_idx * gridDim.y * d));

            // printf("K and V access: %d\n", qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x);
        }
        __syncthreads();

        for (int tile_idx_Q = 0; tile_idx_Q < num_tiles_Q; tile_idx_Q++)
        {

            for (int x = 0; x < d; x++)
            {
                Qi[(thread_idx * d) + x] = Q[qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x];
                // printf("Head %d: Query[%d] = %f\n", head_idx, qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x,  Qi[(thread_idx * d) + x]);

                // printf("Q access: %d\n", qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x);
            }

            float prev_m = m[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx];
            float prev_l = l[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx];
            // printf("m and l access: %d\n", lm_offset + (block_size_Q * tile_idx_Q) + thread_idx);

            float row_max = -INFINITY;
            for (int y = 0; y < block_size_K; y++)
            {
                float sum = 0;
                for (int x = 0; x < d; x++)
                {
                    sum += Qi[(thread_idx * d) + x] * Kj[(y * d) + x];
                    // printf("For index: %d, Head %d: Query[%d] = %f, Kj[%d] = %f\n", (block_size_K * thread_idx) + y, head_idx, (thread_idx * d) + x,  Qi[(thread_idx * d) + x], (y * d) + x,  Kj[(y * d) + x]);
                }
                // printf("sum = %f\n", sum);
                sum *= softmax_scale;
                // printf("sum after scaling = %f\n", sum);
                S[(block_size_K * thread_idx) + y] = sum;
                // printf("Head %d: S[%d] = %f\n",  head_idx, (block_size_K * thread_idx) + y, S[(block_size_K * thread_idx) + y] );

                if (sum > row_max)
                    row_max = sum;
            }

            float row_sum = 0;
            for (int y = 0; y < block_size_K; y++)
            {
                S[(block_size_K * thread_idx) + y] = __expf(S[(block_size_K * thread_idx) + y] - row_max);
                row_sum += S[(block_size_K * thread_idx) + y];
            }

            float new_m = fmax(prev_m, row_max);
            float new_l = (__expf(prev_m - new_m) * prev_l) + (__expf(row_max - new_m) * row_sum);

            for (int x = 0; x < d; x++)
            {
                float weighted_sum = 0;
                for (int y = 0; y < block_size_K; y++)
                {
                    weighted_sum += S[(block_size_K * thread_idx) + y] * Vj[(y * d) + x];
                }
                __syncthreads();
                int index = qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x;
                O[index] = (1 / new_l) *
                           ((prev_l * __expf(prev_m - new_m) * O[index]) +
                            (__expf(row_max - new_m) * weighted_sum));
                // printf("O access after update: %d\n", index);
            }

            m[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx] = new_m;
            l[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx] = new_l;
        }
        __syncthreads();
    }
}


__global__ void backward_kernel(const float *Q, const float *K, const float *V, const int N, const int d,
                               const int num_tiles_K, const int num_tiles_Q, const int block_size_K, const int block_size_Q,
                               const float softmax_scale, float *l, float *m, float *O,
                               float* dQ, float* dK, float* dV, float* dO
                               )
{
    int thread_idx = threadIdx.x;
    int batch_idx = blockIdx.x;
    int head_idx = blockIdx.y;

    int qkv_offset = (batch_idx * gridDim.y * N * d) + (head_idx * N * d);
    int lm_offset = (batch_idx * gridDim.y * N) + (head_idx * N);

    extern __shared__ float shared_memory[];
    int tile_size = block_size_K * d;

    float *Qi = shared_memory;
    float *Kj = &shared_memory[tile_size];
    float *Vj = &shared_memory[tile_size * 2];
   
    float* dKj = &shared_memory[tile_size * 3];
    float* dVj = &shared_memory[tile_size * 4];
    float* Oi = &shared_memory[tile_size * 5];
    float* dOi = &shared_memory[tile_size * 6];

    float *S = &shared_memory[tile_size * 7];
    float *dS = &shared_memory[tile_size * 7 + block_size_K * block_size_Q];



    for (int tile_idx_K = 0; tile_idx_K < num_tiles_K; tile_idx_K++)
    {
        for (int x = 0; x < d; x++)
        {
            // printf("K VALUE: %f\n", K[qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x]);
            int hbm_idx = qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x;
            int sram_idx = (thread_idx * d) + x;

            Kj[sram_idx] = K[hbm_idx];
            Vj[sram_idx] = V[hbm_idx];
            dKj[sram_idx] = 0.0f;
            dVj[sram_idx] = 0.0f;
            // printf("Head %d: Key[%d] = %f, Value[%d] = %f, qkv_offset = %d\n", head_idx, qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x, Kj[(thread_idx * d) + x],
            // qkv_offset + (tile_size * tile_idx_K) + (thread_idx * gridDim.y * d) + x, Vj[(thread_idx * d) + x], qkv_offset);
            // printf("ADDITIONAL ON TOP OF OFFSET = %d\n",   (thread_idx * gridDim.y * d));

            // printf("K and V access: %d\n", qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x);
        }
        


        for (int tile_idx_Q = 0; tile_idx_Q < num_tiles_Q; tile_idx_Q++)
        {

            for (int x = 0; x < d; x++)
            {
                int hbm_idx = qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x;
                int sram_idx = (thread_idx * d) + x;

                Qi[sram_idx] = Q[hbm_idx];
                Oi[sram_idx] = O[hbm_idx];
                dOi[sram_idx] = dO[hbm_idx];
                // printf("Head %d: Query[%d] = %f\n", head_idx, qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x,  Qi[(thread_idx * d) + x]);

                // printf("Q access: %d\n", qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x);
            }

            float m_val = m[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx];
            float l_val = l[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx];
            // printf("m and l access: %d\n", lm_offset + (block_size_Q * tile_idx_Q) + thread_idx);

            for (int y = 0; y < block_size_K; y++)
            {
                float sum = 0;
                for (int x = 0; x < d; x++)
                {
                    sum += Qi[(thread_idx * d) + x] * Kj[(y * d) + x];
                    // printf("For index: %d, Head %d: Query[%d] = %f, Kj[%d] = %f\n", (block_size_K * thread_idx) + y, head_idx, (thread_idx * d) + x,  Qi[(thread_idx * d) + x], (y * d) + x,  Kj[(y * d) + x]);
                }
                // printf("sum = %f\n", sum);
                sum *= softmax_scale;
                // printf("sum after scaling = %f\n", sum);
                S[(block_size_K * thread_idx) + y] = sum;
                // printf("Head %d: S[%d] = %f\n",  head_idx, (block_size_K * thread_idx) + y, S[(block_size_K * thread_idx) + y] );
            }

            // float row_sum = 0;
            for (int y = 0; y < block_size_K; y++)
            {   
                // printf("l value: %f\n", l_val);
                S[(block_size_K * thread_idx) + y] = (1 / l_val) * __expf(S[(block_size_K * thread_idx) + y] - m_val);
                // printf("S VALUE: %f\n", S[(block_size_K * thread_idx) + y]);
            }
            __syncthreads();

            for (int x = 0; x < d; x++) {
                float sum = 0;
                for (int y = 0; y < block_size_K; y++) {
                    sum += S[(block_size_K * y) + thread_idx] * dOi[(thread_idx * d) + x];
                }
                dVj[(thread_idx * d) + x] += sum;
            }

            for (int y = 0; y < block_size_K; y++) {
                float sum = 0;
                for (int x = 0; x < d; x++) {
                    sum += dOi[(thread_idx * d) + x] * Vj[(y * d) + x];
                }
                dS[(block_size_K * thread_idx) + y] = sum;
            }

            float Di = 0;
            for (int x = 0; x < d; x++) {
                Di += dOi[(thread_idx * d) + x] * Oi[(thread_idx * d) + x];
            }

            for (int y = 0; y < block_size_K; ++y) {
                dS[(block_size_K * thread_idx) + y] = S[(block_size_K * thread_idx) + y] * (dS[(block_size_K * thread_idx) + y] - Di);
            }

            for (int x = 0; x < d; x++) {
                float sum = 0;
                for (int y = 0; y < block_size_K; y++) {
                    sum += dS[(block_size_K * thread_idx) + y] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                dQ[qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x] += sum;
            }

            for (int x = 0; x < d; x++) {
                float sum = 0;
                for (int y = 0; y < block_size_K; y++) {
                    sum += dS[(block_size_K * y) + thread_idx] * Qi[(y * d) + x];
                }
                sum *= softmax_scale;
                dKj[(thread_idx * d) + x] += sum;
            }
        }

        for (int x = 0; x < d; x++) {
            dK[qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x] = dKj[(thread_idx * d) + x];
            dV[qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x] = dVj[(thread_idx * d) + x];
        }
    
    }
}

extern "C"
{
    void launch_flashattention_forward(float *Q, float *K, float *V, float *O, float *l, float *m, int batch_size, int num_heads, int N, int d)
    {

        int block_size_K, block_size_Q;
        int max_sram_size;
        hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

        // TODO: Optimise memory usage for lower d, low hanging fruit for those that can be 4x

        // Assume SRAM >= 32768
        if (d>2048) return;
        // block_size_K =8; block_size_Q =8;


        block_size_K = min(min(N, 2048/d), 64); block_size_Q = min(min(N, 2048/d), 64);
        while(N % block_size_K != 0){
            block_size_K/=2;
            block_size_Q/=2;
        }


        const int num_tiles_K = ceil((float)N / block_size_K);
        const int num_tiles_Q = ceil((float)N / block_size_Q);

        const float softmax_scale = 1.0 / sqrt(d);

        float *d_Q, *d_K, *d_V, *d_O, *d_l, *d_m;
        const int Q_size = batch_size * num_heads * N * d;
        const int l_size = batch_size * num_heads * N;

        hipMalloc((void **)&d_Q, Q_size * sizeof(float));
        hipMalloc((void **)&d_K, Q_size * sizeof(float));
        hipMalloc((void **)&d_V, Q_size * sizeof(float));
        hipMalloc((void **)&d_O, Q_size * sizeof(float));
        hipMalloc((void **)&d_l, l_size * sizeof(float));
        hipMalloc((void **)&d_m, l_size * sizeof(float));

        hipMemcpy(d_Q, Q, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_K, K, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_V, V, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_O, O, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_l, l, l_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_m, m, l_size * sizeof(float), hipMemcpyHostToDevice);

        const int shared_mem_size = (2 * block_size_K * d * sizeof(float)) + (block_size_Q * d * sizeof(float)) + (block_size_K * block_size_Q * sizeof(float)) + (2 * block_size_K * sizeof(float));

        // Uncomment if you want to see SRAM requested
        // printf("FORWARD: d: %d | num_head: %d | N: %d | Block Size Q: %d | Block Size K: %d | Max shared memory: %d, requested shared memory: %d \n", d, num_heads, N, block_size_Q, block_size_K, max_sram_size, shared_mem_size);

        // // This should never run if your code checks for too large d
        // if (shared_mem_size >= max_sram_size) {
        //     // fprintf(stderr, "Too much SRAM requested: %f\n", shared_mem_size);
        //     // exit(EXIT_FAILURE);
        //     // printf("Too much SRAM requested: %d\n", shared_mem_size);
        //     return;
        // }

        dim3 grid_dim(batch_size, num_heads);
        dim3 block_dim(block_size_K);

        // Launch the kernel
        forward_kernel<<<grid_dim, block_dim, shared_mem_size>>>(
            d_Q, d_K, d_V, N, d, num_tiles_K, num_tiles_Q, block_size_K, block_size_Q, softmax_scale, d_l, d_m, d_O);
        hipDeviceSynchronize();

        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Error in kernel launch: %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        hipMemcpy(O, d_O, Q_size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(m, d_m, l_size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(l, d_l, l_size * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_Q);
        hipFree(d_K);
        hipFree(d_V);
        hipFree(d_O);
        hipFree(d_l);
        hipFree(d_m);
    }
}

int find_max_block_size(int d, int max_sram_size) {
    int B = 1;
    int best_B = 1;

    while (true) {
        size_t shared_mem_size = sizeof(float) * (
            (7 * B * d) +         // Qi, Kj, Vj, dKj, dVj, Oi, dOi
            (2 * B * B) +         // S, dS
            (2 * B)               // l, m if needed
        );

        if (shared_mem_size > max_sram_size)
            break;

        best_B = B;
        B *= 2;  // Try next power of 2, up to 64
        if (B > 64) break;
    }

    return best_B;
}

extern "C"
{
    void launch_flashattention_backward(float *Q, float *K, float *V, float *O, float *dQ,  float *dK,  float *dV, float *dO,  float *l, float *m, int batch_size, int num_heads, int N, int d)
    {

        int block_size_K, block_size_Q;
        int max_sram_size;
        hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

        // TODO: Optimise memory usage for lower d, low hanging fruit for those that can be 4x

        // Assume SRAM >= 32768
        if (d>2048) return;

        block_size_K = min(find_max_block_size(d, max_sram_size), N); block_size_Q = block_size_K;
        // block_size_K = 4; block_size_Q = 4;
        // block_size_K = min(min(d, 2048/d), 64); block_size_Q = min(min(d, 2048/d), 64);
        

        const int num_tiles_K = ceil((float)N / block_size_K);
        const int num_tiles_Q = ceil((float)N / block_size_Q);

        const float softmax_scale = 1.0 / sqrt(d);

        float *d_Q, *d_dQ, *d_K, *d_dK, *d_V, *d_dV, *d_O, *d_dO, *d_l, *d_m;
        const int Q_size = batch_size * num_heads * N * d;
        const int l_size = batch_size * num_heads * N;

        hipMalloc((void **)&d_Q, Q_size * sizeof(float));
        hipMalloc((void **)&d_dQ, Q_size * sizeof(float));

        hipMalloc((void **)&d_K, Q_size * sizeof(float));
        hipMalloc((void **)&d_dK, Q_size * sizeof(float));

        hipMalloc((void **)&d_V, Q_size * sizeof(float));
        hipMalloc((void **)&d_dV, Q_size * sizeof(float));

        hipMalloc((void **)&d_O, Q_size * sizeof(float));
        hipMalloc((void **)&d_dO, Q_size * sizeof(float));
        
        hipMalloc((void **)&d_l, l_size * sizeof(float));
        hipMalloc((void **)&d_m, l_size * sizeof(float));

        hipMemcpy(d_Q, Q, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_dQ, dQ, Q_size * sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(d_K, K, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_dK, dK, Q_size * sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(d_V, V, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_dV, dV, Q_size * sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(d_O, O, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_dO, dO, Q_size * sizeof(float), hipMemcpyHostToDevice);
        
        hipMemcpy(d_l, l, l_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_m, m, l_size * sizeof(float), hipMemcpyHostToDevice);

        const int shared_mem_size = (4 * block_size_K * d * sizeof(float)) + (3 * block_size_Q * d * sizeof(float)) + (2 * block_size_K * block_size_Q * sizeof(float)) + (2 * block_size_K * sizeof(float));;

        // Uncomment if you want to see SRAM requested
        printf("BACKWARD: d: %d | num_head: %d | N: %d | Block Size Q: %d | Block Size K: %d | Max shared memory: %d, requested shared memory: %d \n", d, num_heads, N, block_size_Q, block_size_K, max_sram_size, shared_mem_size);

        // // This should never run if your code checks for too large d
        if (shared_mem_size >= max_sram_size) {
            printf("Too much SRAM requested: %d\n", shared_mem_size);
            return;
        }

        dim3 grid_dim(batch_size, num_heads);
        dim3 block_dim(block_size_K);

        // Launch the kernel
        backward_kernel<<<grid_dim, block_dim, shared_mem_size>>>(
            d_Q, d_K, d_V, N, d, num_tiles_K, num_tiles_Q, block_size_K, block_size_Q, softmax_scale, d_l, d_m, d_O, d_dQ, d_dK, d_dV, d_dO);
        hipDeviceSynchronize();

        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Error in kernel launch: %s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        hipMemcpy(dO, d_dO, Q_size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(dQ, d_dQ, Q_size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(dK, d_dK, Q_size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(dV, d_dV, Q_size * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_Q);
        hipFree(d_dQ);
        hipFree(d_K);
        hipFree(d_dK);
        hipFree(d_V);
        hipFree(d_dV);
        hipFree(d_O);
        hipFree(d_dO);
        hipFree(d_l);
        hipFree(d_m);
    }
}

__global__ void forward_kernel_causal(const float *Q, const float *K, const float *V, const int N, const int d,
                                      const int num_tiles_K, const int num_tiles_Q, const int block_size_K, const int block_size_Q,
                                      const float softmax_scale, float *l, float *m, float *O)
{

    int thread_idx = threadIdx.x;
    int batch_idx = blockIdx.x;
    int head_idx = blockIdx.y;

    int qkv_offset = (batch_idx * gridDim.y * N * d) + (head_idx * N * d);
    int lm_offset = (batch_idx * gridDim.y * N) + (head_idx * N);

    extern __shared__ float shared_memory[];
    int tile_size = block_size_K * d;
    float *Qi = shared_memory;
    float *Kj = &shared_memory[tile_size];
    float *Vj = &shared_memory[tile_size * 2];
    float *S = &shared_memory[tile_size * 3];


    for (int tile_idx_K = 0; tile_idx_K < num_tiles_K; tile_idx_K++)
    {
        // Load Kj, Vj into SRAM, each thread loads Kj+thread_idx, Vj+thread_idx concurrently
        for (int x = 0; x < d; x++)
        {
            Kj[(thread_idx * d) + x] = K[qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x];
            Vj[(thread_idx * d) + x] = V[qkv_offset + (tile_size * tile_idx_K) + (thread_idx * d) + x];
        }
        __syncthreads();

        for (int tile_idx_Q = 0; tile_idx_Q < num_tiles_Q; tile_idx_Q++)
        {
            // Coarse-grained optimisation (Block-level)
            if (tile_idx_Q < tile_idx_K) continue;
            
            // Load Qi into SRAM, each thread loads Qi+thread_idx concurrently
            for (int x = 0; x < d; x++)
            {
                Qi[(thread_idx * d) + x] = Q[qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x];
            }

            // Load li, mi into SRAM
            float prev_m = m[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx];
            float prev_l = l[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx];

            // Compute Sij & rowmax
            float row_max = -INFINITY;
            for (int y = 0; y < block_size_K; y++)
            {
                float sum = 0;
                for (int x = 0; x < d; x++)
                {
                    sum += Qi[(thread_idx * d) + x] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                S[(block_size_K * thread_idx) + y] = sum;

                if (sum > row_max)
                    row_max = sum;
            }

            // Compute rowsum and Pij + apply fine-grained mask (element-level)
            float row_sum = 0;
            for (int y = 0; y < block_size_K; y++)
            {
                if ((tile_idx_Q * block_size_Q + thread_idx) >= (tile_idx_K * block_size_K) + y) {
                    S[(block_size_K * thread_idx) + y] = __expf(S[(block_size_K * thread_idx) + y] - row_max);
                } else {
                    S[(block_size_K * thread_idx) + y] = 0.0;
                }

                row_sum += S[(block_size_K * thread_idx) + y];
            }

            // Compute m_new, l_new
            float new_m = fmax(prev_m, row_max);
            float new_l = (__expf(prev_m - new_m) * prev_l) + (__expf(row_max - new_m) * row_sum);

            // Write O to HBM
            for (int x = 0; x < d; x++)
            {
                float weighted_sum = 0;
                for (int y = 0; y < block_size_K; y++)
                {
                    weighted_sum += S[(block_size_K * thread_idx) + y] * Vj[(y * d) + x];
                }
                __syncthreads();
                int index = qkv_offset + (tile_size * tile_idx_Q) + (thread_idx * d) + x;
                O[index] = (1 / new_l) *
                        ((prev_l * __expf(prev_m - new_m) * O[index]) +
                            (__expf(row_max - new_m) * weighted_sum));
            }

            // Write li, mi to HBM
            m[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx] = new_m;
            l[lm_offset + (block_size_Q * tile_idx_Q) + thread_idx] = new_l;
        }
        __syncthreads();
    }
}

extern "C"
{
    void launch_flashattention_forward_causal(float *Q, float *K, float *V, float *O, float *l, float *m, int batch_size, int num_heads, int N, int d)
    {

        int block_size_K, block_size_Q;

        if (d>2048) return;

        block_size_K = min(min(N, 2048/d), 64); block_size_Q = min(min(N, 2048/d), 64);

        const int num_tiles_K = ceil((float)N / block_size_K);
        const int num_tiles_Q = ceil((float)N / block_size_Q);

        const float softmax_scale = 1.0 / sqrt(d);

        float *d_Q, *d_K, *d_V, *d_O, *d_l, *d_m;
        const int Q_size = batch_size * num_heads * N * d;
        const int l_size = batch_size * num_heads * N;

        hipMalloc((void **)&d_Q, Q_size * sizeof(float));
        hipMalloc((void **)&d_K, Q_size * sizeof(float));
        hipMalloc((void **)&d_V, Q_size * sizeof(float));
        hipMalloc((void **)&d_O, Q_size * sizeof(float));
        hipMalloc((void **)&d_l, l_size * sizeof(float));
        hipMalloc((void **)&d_m, l_size * sizeof(float));


        hipMemcpy(d_Q, Q, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_K, K, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_V, V, Q_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_l, l, l_size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_m, m, l_size * sizeof(float), hipMemcpyHostToDevice);

        const int shared_mem_size = (2 * block_size_K * d * sizeof(float)) + (block_size_Q * d * sizeof(float)) + (block_size_K * block_size_Q * sizeof(float)) + (2 * block_size_K * sizeof(float));

        dim3 grid_dim(batch_size, num_heads);
        dim3 block_dim(block_size_K);

        // Launch the kernel
        forward_kernel_causal<<<grid_dim, block_dim, shared_mem_size>>>(
            d_Q, d_K, d_V, N, d, num_tiles_K, num_tiles_Q, block_size_K, block_size_Q, softmax_scale, d_l, d_m, d_O);
        hipDeviceSynchronize();

        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Error in kernel launch: %s\n", hipGetErrorString(err));
            // exit(EXIT_FAILURE);
            return;
        }

        hipMemcpy(O, d_O, Q_size * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_Q);
        hipFree(d_K);
        hipFree(d_V);
        hipFree(d_O);
        hipFree(d_l);
        hipFree(d_m);
    }
}